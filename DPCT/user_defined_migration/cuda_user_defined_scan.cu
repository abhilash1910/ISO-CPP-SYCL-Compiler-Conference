/
// RUN: dpct -format-range=none -out-root %T/cuda_user_defined_scan cuda_user_defined_scan.cu --cuda-include-path="%cuda-path/include" --usm-level=none --rule-file=cuda_user_defined_scan.yaml -- -x cuda --cuda-host-only

// CHECK: #include <oneapi/dpl/execution>
// CHECK: #include <oneapi/dpl/algorithm>
#include <stddef.h>
#include "user_define_rule_header_order1.h"