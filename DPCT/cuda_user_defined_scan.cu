/
// RUN: dpct -format-range=none -out-root %T/cuda_user_defined_scan cuda_user_defined_scan.cu --cuda-include-path="%cuda-path/include" --usm-level=none --rule-file=cuda_user_defined_scan.yaml -- -x cuda --cuda-host-only

// CHECK: #include <oneapi/dpl/execution>
// CHECK: #include <oneapi/dpl/algorithm>
#include <stddef.h>
#include "cuda_user_defined_scan.h"